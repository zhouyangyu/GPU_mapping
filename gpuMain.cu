#include "hip/hip_runtime.h"
//
//  gpuMain.cpp
//  mapping
//
//  Created by Joe Zhou on 12/13/16.
//  Copyright © 2016 Joe Zhou. All rights reserved.
//

#include <stdio.h>

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include "sa.h"
#include <ctime>

__device__
void strncmpGPU(const char *s1, const char *s2, size_t n, int* result){
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for( ; n>0;s1++, s2++, --n)
        if(*s1 != *s2)
            *result = ((*(unsigned char *)s1 < *(unsigned char *)s2) ? -1 : +1);
        else if (*s1 == '\0')
            *result = 0;
    *result = 0;
}

//__global__
//void saxpy(int n, float a, float *x, float *y)
//{
//    int i = blockIdx.x*blockDim.x + threadIdx.x;
//    if (i < n) y[i] = a*x[i] + y[i];
//}
//
//__global__
//void foo(int* result, int n)
//{
//    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
//    if (threadID < n)
//        result[threadID] = threadID;
//}

__global__
void search(char*** readsGroups, int numReads, char* ref, int* sa, int* result, int m, int n)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    
    char** reads = readsGroups[threadID];
    //int m = strlen(reads[1]);  // get length of pattern, needed for strncmp()
    //int n = strlen(ref);
    
    //int *result = new int[n];
    //int unmatched = 0;
    
    for(int i=0; i< numReads; i++){
        char* pat = reads[i];
        // Do simple binary search for the pat in txt using the
        // built suffix array
        int l = 0, r = n-1;  // Initilize left and right indexes
        while (l <= r)
        {
            // See if 'pat' is prefix of middle suffix in suffix array
            int mid = l + (r - l)/2;
            int *res = new int;
            strncmpGPU(pat, ref+sa[mid], m, res);
            
            // If match found at the middle, print it and return
            if (*res == 0)
            {
                //return sa[mid];
                result[sa[mid]] = result[sa[mid]] + 1;
            }
            
            // Move to left half if pattern is alphabtically less than
            // the mid suffix
            if (*res < 0) r = mid - 1;
            
            // Otherwise move to right half
            else l = mid + 1;
        }
        
        //unmatched++;
    }
}

//Genomefilename, readsFileName, number of blocks, number of threads, number of reads
int main(int argc, const char * argv[]) {
    clock_t start, end;
    start = clock();
    //read sequence from file, allocate sequence on heap b/c large size
    std::string * input = new std::string();
    std::ifstream file;
    file.open(argv[1]);
    getline(file,*input);
    file.close();
    std::cout << "ref file read\n";
    std::cout << "time took = " << (clock()-start)/double(CLOCKS_PER_SEC)*1000 << "\n";
    std::string sequence = *input;
    //delete input;
    sequence += "$";//end of sequence char
    int numBlocks= *argv[3];
    int numThreads = *argv[4];
    int totalThreads = numBlocks * numThreads;
    
    char* s = (char*)malloc(sequence.size()*sizeof(char));//on heap
    std::copy(sequence.begin(), sequence.end(), s);
    s[sequence.size()] = '\0';
    
    std::vector<int> sav = buildSuffixArray(sequence, sequence.size());
    std::cout << "suffix tree build\n";
    std::cout << "time took = " << (clock()-start)/double(CLOCKS_PER_SEC)*1000 << "\n";
    int* sa = (int*)malloc(sav.size()*sizeof(int));//on heap
    sa = &sav[0];
    file.open(argv[2]);
    std::string line;
    
    int numReads = *argv[5];
    int index = 0;
    char** reads = (char**)malloc((numReads+1000000000000000000+1)*sizeof(char*));
    std::cout << "2 \n";
    while(std::getline(file, line)){
        //convert string to char* for checkForSubString();
        char* currRead = (char*)malloc(line.size()+1);
        memcpy(currRead, line.c_str(),line.size());
        reads[index] = currRead;
        //free(currRead);
        index = index +1;
    }
    std::cout << "3 \n";
    char*** readsGroups = (char***)malloc(totalThreads*sizeof(char**));
    int groupsize = numReads/totalThreads;
    for(int i=0;i<totalThreads;i++){
        char** readg = (char**)malloc(groupsize*sizeof(char*));
        memcpy(readg, reads+i*groupsize, groupsize*sizeof(char*));
        readsGroups[i] = readg;
    }
    
    std::cout << "reads read \n";
    std::cout << "time took = " << (clock()-start)/double(CLOCKS_PER_SEC)*1000 << "\n";
    
    int* result = (int*)malloc(sequence.size()*sizeof(int));
    for(int i=0; i<sequence.size(); i++){
        result[i] = 0;
    }
    
    //allocate GPU memory
    char* d_s; int* d_sa; char*** d_readsGroups; int* d_result;
    hipMalloc(&d_s, sequence.size()*sizeof(char));
    hipMalloc(&d_sa, sav.size()*sizeof(int));
    hipMalloc(&d_readsGroups, totalThreads*sizeof(char**));
    hipMalloc(&d_result, sequence.size()*sizeof(int));
    
    //transfer data from CPU to GPU
    hipMemcpy(d_s,s,sequence.size()*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_sa,sa,sav.size()*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_readsGroups,readsGroups,totalThreads*sizeof(char**),hipMemcpyHostToDevice);
    hipMemcpy(d_result,result,sequence.size()*sizeof(int),hipMemcpyHostToDevice);
    
    std::cout << "performing search on GPU\n";
    std::cout << "time took = " << (clock()-start)/double(CLOCKS_PER_SEC)*1000 << "\n";
    
    //perform search
    search<<<numBlocks,numThreads>>>(d_readsGroups, numReads, d_s, d_sa, d_result, strlen(reads[1]), strlen(s));
    //foo<<<256,256>>>(d_result, strlen(s));
    
    std::cout << "search finished\n";
    std::cout << "time took = " << (clock()-start)/double(CLOCKS_PER_SEC)*1000 << "\n";
    
    //transfer alignment result back to CPU
    hipMemcpy(result, d_result, sequence.size()*sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    
    //free heap memory on GPU and CPU
        hipFree(d_s);
        hipFree(d_sa);
        hipFree(d_readsGroups);
        hipFree(d_result);
        free(s);
        free(sa);
    for(int i=0; i < numReads;i++){
        //free(reads[i]);
    }
    
    //    for(int i=0; i<sequence.size();i++){
    //        std::cout << result[i] << " ";
    //    }
    
    //free(result);
    
    end = clock();
    std::cout << "finished";
    std::cout << "time took = " << (end-start)/double(CLOCKS_PER_SEC)*1000 << "\n";
}