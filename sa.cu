// C++ program for building LCP array for given text

#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <iostream>
#include <algorithm>


// Structure to store information of a suffix
struct suffix
{
    int index;  // To store original index
    int rank[2]; // To store ranks and next rank pair
};

// A comparison function used by sort() to compare two suffixes
// Compares two pairs, returns 1 if first pair is smaller
int cmp(struct suffix a, struct suffix b)
{
    return (a.rank[0] == b.rank[0])? (a.rank[1] < b.rank[1] ?1: 0):
    (a.rank[0] < b.rank[0] ?1: 0);
}

// This is the main function that takes a string 'txt' of size n as an
// argument, builds and return the suffix array for the given string
std::vector<int> buildSuffixArray(std::string txt, int n)
{
    // A structure to store suffixes and their indexes
    struct suffix suffixes[n];
    
    // Store suffixes and their indexes in an array of structures.
    // The structure is needed to sort the suffixes alphabatically
    // and maintain their old indexes while sorting
    for (int i = 0; i < n; i++)
    {
        suffixes[i].index = i;
        suffixes[i].rank[0] = txt[i] - 'a';
        suffixes[i].rank[1] = ((i+1) < n)? (txt[i + 1] - 'a'): -1;
    }
    
    // Sort the suffixes using the comparison function
    // defined above.
    std::sort(suffixes, suffixes+n, cmp);
    
    // At his point, all suffixes are sorted according to first
    // 2 characters.  Let us sort suffixes according to first 4
    // characters, then first 8 and so on
    int ind[n];  // This array is needed to get the index in suffixes[]
    // from original index.  This mapping is needed to get
    // next suffix.
    for (int k = 4; k < 2*n; k = k*2)
    {
        // Assigning rank and index values to first suffix
        int rank = 0;
        int prev_rank = suffixes[0].rank[0];
        suffixes[0].rank[0] = rank;
        ind[suffixes[0].index] = 0;
        
        // Assigning rank to suffixes
        for (int i = 1; i < n; i++)
        {
            // If first rank and next ranks are same as that of previous
            // suffix in array, assign the same new rank to this suffix
            if (suffixes[i].rank[0] == prev_rank &&
                suffixes[i].rank[1] == suffixes[i-1].rank[1])
            {
                prev_rank = suffixes[i].rank[0];
                suffixes[i].rank[0] = rank;
            }
            else // Otherwise increment rank and assign
            {
                prev_rank = suffixes[i].rank[0];
                suffixes[i].rank[0] = ++rank;
            }
            ind[suffixes[i].index] = i;
        }
        
        // Assign next rank to every suffix
        for (int i = 0; i < n; i++)
        {
            int nextindex = suffixes[i].index + k/2;
            suffixes[i].rank[1] = (nextindex < n)?
            suffixes[ind[nextindex]].rank[0]: -1;
        }
        
        // Sort the suffixes according to first k characters
        std::sort(suffixes, suffixes+n, cmp);
    }
    
    // Store indexes of all sorted suffixes in the suffix array
    std::vector<int>suffixArr;
    for (int i = 0; i < n; i++)
        suffixArr.push_back(suffixes[i].index);
    
    // Return the suffix array
    return  suffixArr;
}

// A suffix array based search function to search a given pattern
// 'pat' in given text 'txt' using suffix array suffArr[]
//int search(char *pat, char *txt, int *suffArr)
//{
//    int m = strlen(pat);  // get length of pattern, needed for strncmp()
//    int n = strlen(txt);
//    
//    // Do simple binary search for the pat in txt using the
//    // built suffix array
//    int l = 0, r = n-1;  // Initilize left and right indexes
//    while (l <= r)
//    {
//        // See if 'pat' is prefix of middle suffix in suffix array
//        int mid = l + (r - l)/2;
//        int res = strncmp(pat, txt+suffArr[mid], m);
//        
//        // If match found at the middle, print it and return
//        if (res == 0)
//        {
//            return suffArr[mid];
//        }
//        
//        // Move to left half if pattern is alphabtically less than
//        // the mid suffix
//        if (res < 0) r = mid - 1;
//        
//        // Otherwise move to right half
//        else l = mid + 1;
//    }
//    
//    // We reach here if return statement in loop is not executed
//    return -1;
//}

// Utility function to print an array
void printArr(std::vector<int>arr, int n)
{
    for (int i = 0; i < n; i++)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

// Driver program
//int main()
//{
//    string str = "banana";
//    
//    vector<int>suffixArr = buildSuffixArray(str, str.length());
//    int n = suffixArr.size();
//    
//    cout << "Suffix Array : \n";
//    printArr(suffixArr, n);
//    
//    return 0;
//}